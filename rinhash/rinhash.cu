#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <stdint.h>
#include <stdio.h>
#include <string.h>
#include <vector>
#include <stdexcept>

// Include shared device functions (chỉ include .cuh hoặc .h)
#include "rinhash_device.cuh"
#include "argon2d_device.cuh"
#include "sha3-256.cu"
#include "blake3_device.cuh"

// 🚀 GTX 1060 3GB OPTIMIZED: Balance memory usage vs performance
#define MAX_BATCH_BLOCKS 32768

// Kernel đơn: mỗi lần chỉ chạy 1 thread
extern "C" __global__ void rinhash_cuda_kernel(
    const uint8_t* input, 
    size_t input_len, 
    uint8_t* output,
    block* memory,      // bộ nhớ argon2 đã cấp phát trên host, truyền vào
    uint32_t m_cost
) {
    // Chỉ 1 thread xử lý
    if (threadIdx.x == 0) {
        uint8_t blake3_out[32];
        light_hash_device(input, input_len, blake3_out);

        uint8_t salt[11] = { 'R','i','n','C','o','i','n','S','a','l','t' };
        uint8_t argon2_out[32];
        device_argon2d_hash(argon2_out, blake3_out, 32, 2, m_cost, 1, memory, salt, sizeof(salt));

        uint8_t sha3_out[32];
        sha3_256_device(argon2_out, 32, sha3_out);

        // Copy kết quả ra output
        for (int i = 0; i < 32; i++) output[i] = sha3_out[i];
    }
}

// 🚀 OPTIMIZED Kernel batch with target-aware early termination
extern "C" __global__ void rinhash_cuda_kernel_batch(
    const uint8_t* headers,         // num_blocks * 80 bytes
    size_t header_len,              // = 80
    uint8_t* outputs,               // num_blocks * 32 bytes
    uint32_t num_blocks,
    block* memories,                // num_blocks * m_cost * sizeof(block)
    uint32_t m_cost
) {
    uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= num_blocks) return;
    
    const uint8_t* input = headers + tid * header_len;
    uint8_t* output = outputs + tid * 32;
    block* memory = memories + tid * m_cost;

    uint8_t blake3_out[32];
    light_hash_device(input, header_len, blake3_out);

    uint8_t salt[11] = { 'R','i','n','C','o','i','n','S','a','l','t' };
    uint8_t argon2_out[32];
    device_argon2d_hash(argon2_out, blake3_out, 32, 2, m_cost, 1, memory, salt, sizeof(salt));

    sha3_256_device(argon2_out, 32, output);
}

// 🚀 NEW: Target-aware kernel with atomic solution detection
extern "C" __global__ void rinhash_cuda_kernel_optimized(
    const uint8_t* headers,
    size_t header_len,
    uint8_t* outputs,
    uint32_t num_blocks,
    block* memories,
    uint32_t m_cost,
    uint32_t* target,           // 8 x uint32_t target
    uint32_t* solution_found,   // atomic flag
    uint32_t* solution_nonce    // winning nonce
) {
    uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= num_blocks) return;
    
    // Early exit if solution already found
    if (atomicAdd(solution_found, 0) > 0) return;
    
    const uint8_t* input = headers + tid * header_len;
    uint8_t* output = outputs + tid * 32;
    block* memory = memories + tid * m_cost;

    uint8_t blake3_out[32];
    light_hash_device(input, header_len, blake3_out);

    uint8_t salt[11] = { 'R','i','n','C','o','i','n','S','a','l','t' };
    uint8_t argon2_out[32];
    device_argon2d_hash(argon2_out, blake3_out, 32, 2, m_cost, 1, memory, salt, sizeof(salt));

    sha3_256_device(argon2_out, 32, output);
    
    // Quick target check - convert hash to uint32_t array
    uint32_t* hash_words = (uint32_t*)output;
    
    // Check if hash meets target (little-endian comparison from back)
    bool meets_target = true;
    for (int i = 7; i >= 0; i--) {
//        uint32_t* hash_words[i] = ((hash_words[i] & 0xFF) << 24) | 
//                               ((hash_words[i] & 0xFF00) << 8) | 
//                               ((hash_words[i] & 0xFF0000) >> 8) | 
//                               ((hash_words[i] & 0xFF000000) >> 24);
        if (hash_words[i] > target[i]) {
            meets_target = false;
            break;
        } else if (hash_words[i] < target[i]) {
            break; // This hash is better, continue to set solution
        }
    }
    
    if (meets_target) {
        // Atomic solution detection - first thread wins
        if (atomicCAS(solution_found, 0, 1) == 0) {
            // Extract nonce from header (last 4 bytes)
            uint32_t* header_words = (uint32_t*)(input);
            *solution_nonce = header_words[19]; // nonce is at offset 76 bytes = word 19
        }
    }
}


// Helper: kiểm tra lỗi CUDA
inline void check_cuda(const char* msg) {
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA error at %s: %s\n", msg, hipGetErrorString(err));
        throw std::runtime_error("CUDA error");
    }
}

// Cleanup persistent GPU memory (required by rinhash_scanhash.cpp)
extern "C" void rinhash_cuda_cleanup_persistent() {
    // Reset CUDA device to clean up any persistent memory
    hipDeviceReset();
}

// RinHash CUDA implementation (single)
extern "C" void rinhash_cuda(const uint8_t* input, size_t input_len, uint8_t* output) {
    uint8_t *d_input = nullptr;
    uint8_t *d_output = nullptr;
    block* d_memory = nullptr;
    uint32_t m_cost = 64;

    hipError_t err;

    // Alloc device memory
    err = hipMalloc(&d_input, input_len);
    if (err != hipSuccess) { fprintf(stderr, "CUDA: alloc input fail\n"); return; }

    err = hipMalloc(&d_output, 32);
    if (err != hipSuccess) { fprintf(stderr, "CUDA: alloc output fail\n"); hipFree(d_input); return; }

    err = hipMalloc(&d_memory, m_cost * sizeof(block));
    if (err != hipSuccess) { fprintf(stderr, "CUDA: alloc argon2 memory fail\n"); hipFree(d_input); hipFree(d_output); return; }

    // Copy input
    err = hipMemcpy(d_input, input, input_len, hipMemcpyHostToDevice);
    if (err != hipSuccess) { fprintf(stderr, "CUDA: copy input fail\n"); hipFree(d_input); hipFree(d_output); hipFree(d_memory); return; }

    // Launch kernel
    rinhash_cuda_kernel<<<1, 1>>>(d_input, input_len, d_output, d_memory, m_cost);
    hipDeviceSynchronize();
    check_cuda("rinhash_cuda_kernel");

    // Copy result
    err = hipMemcpy(output, d_output, 32, hipMemcpyDeviceToHost);
    if (err != hipSuccess) { fprintf(stderr, "CUDA: copy output fail\n"); }

    // Free
    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_memory);
}

// 🚀 OPTIMIZED: Target-aware batch processing for faster mining
extern "C" void rinhash_cuda_batch_optimized(
    const uint8_t* block_headers,
    size_t block_header_len,
    uint8_t* outputs,
    uint32_t num_blocks,
    uint32_t* target,           // Target for early termination
    uint32_t* solution_found,   // Output: 1 if solution found
    uint32_t* solution_nonce    // Output: winning nonce
) {
    if (num_blocks > MAX_BATCH_BLOCKS) {
        fprintf(stderr, "Batch too large (max %u)\n", MAX_BATCH_BLOCKS);
        return;
    }

    uint8_t *d_headers = nullptr, *d_outputs = nullptr;
    block* d_memories = nullptr;
    uint32_t *d_target = nullptr, *d_solution_found = nullptr, *d_solution_nonce = nullptr;
    uint32_t m_cost = 64;
    
    size_t headers_size = block_header_len * num_blocks;
    size_t outputs_size = 32 * num_blocks;
    size_t memories_size = num_blocks * m_cost * sizeof(block);

    // 🚀 GTX 1060 OPTIMIZED: Define thread configuration first
    const int threads_per_block = 256;
    int blocks = (num_blocks + threads_per_block - 1) / threads_per_block;

    // Allocate GPU memory
    hipError_t err;
    err = hipMalloc(&d_headers, headers_size);
    if (err != hipSuccess) { fprintf(stderr, "CUDA: alloc headers fail\n"); return; }
    err = hipMalloc(&d_outputs, outputs_size);
    if (err != hipSuccess) { fprintf(stderr, "CUDA: alloc outputs fail\n"); hipFree(d_headers); return; }
    err = hipMalloc(&d_memories, memories_size);
    if (err != hipSuccess) { fprintf(stderr, "CUDA: alloc argon2 memories fail\n"); hipFree(d_headers); hipFree(d_outputs); return; }
    err = hipMalloc(&d_target, 8 * sizeof(uint32_t));
    if (err != hipSuccess) { fprintf(stderr, "CUDA: alloc target fail\n"); goto cleanup; }
    err = hipMalloc(&d_solution_found, sizeof(uint32_t));
    if (err != hipSuccess) { fprintf(stderr, "CUDA: alloc solution_found fail\n"); goto cleanup; }
    err = hipMalloc(&d_solution_nonce, sizeof(uint32_t));
    if (err != hipSuccess) { fprintf(stderr, "CUDA: alloc solution_nonce fail\n"); goto cleanup; }

    // Initialize data
    hipMemset(d_outputs, 0xee, outputs_size);
    hipMemset(d_solution_found, 0, sizeof(uint32_t));
    hipMemcpy(d_headers, block_headers, headers_size, hipMemcpyHostToDevice);
    hipMemcpy(d_target, target, 8 * sizeof(uint32_t), hipMemcpyHostToDevice);
    
    rinhash_cuda_kernel_optimized<<<blocks, threads_per_block>>>(
        d_headers, block_header_len, d_outputs, num_blocks, d_memories, m_cost,
        d_target, d_solution_found, d_solution_nonce
    );
    hipDeviceSynchronize();
    check_cuda("rinhash_cuda_kernel_optimized");

    // Copy results back
    err = hipMemcpy(outputs, d_outputs, outputs_size, hipMemcpyDeviceToHost);
    if (err != hipSuccess) { fprintf(stderr, "CUDA: copy output batch fail\n"); }
    
    err = hipMemcpy(solution_found, d_solution_found, sizeof(uint32_t), hipMemcpyDeviceToHost);
    if (err != hipSuccess) { fprintf(stderr, "CUDA: copy solution_found fail\n"); }
    
    err = hipMemcpy(solution_nonce, d_solution_nonce, sizeof(uint32_t), hipMemcpyDeviceToHost);
    if (err != hipSuccess) { fprintf(stderr, "CUDA: copy solution_nonce fail\n"); }

cleanup:
    hipFree(d_headers);
    hipFree(d_outputs);
    hipFree(d_memories);
    hipFree(d_target);
    hipFree(d_solution_found);
    hipFree(d_solution_nonce);
}

// Batch processing version for mining (legacy - kept for compatibility)
extern "C" void rinhash_cuda_batch(
    const uint8_t* block_headers,
    size_t block_header_len,
    uint8_t* outputs,
    uint32_t num_blocks
) {
    if (num_blocks > MAX_BATCH_BLOCKS) {
        fprintf(stderr, "Batch too large (max %u)\n", MAX_BATCH_BLOCKS);
        return;
    }

    uint8_t *d_headers = nullptr, *d_outputs = nullptr;
    block* d_memories = nullptr;
    uint32_t m_cost = 64;
    size_t headers_size = block_header_len * num_blocks;
    size_t outputs_size = 32 * num_blocks;
    size_t memories_size = num_blocks * m_cost * sizeof(block);

    hipError_t err;
    err = hipMalloc(&d_headers, headers_size);
    if (err != hipSuccess) { fprintf(stderr, "CUDA: alloc headers fail\n"); return; }
    err = hipMalloc(&d_outputs, outputs_size);
    if (err != hipSuccess) { fprintf(stderr, "CUDA: alloc outputs fail\n"); hipFree(d_headers); return; }
    err = hipMalloc(&d_memories, memories_size);
    if (err != hipSuccess) { fprintf(stderr, "CUDA: alloc argon2 memories fail\n"); hipFree(d_headers); hipFree(d_outputs); return; }

    hipMemset(d_outputs, 0xee, outputs_size);
    hipMemcpy(d_headers, block_headers, headers_size, hipMemcpyHostToDevice);

    // 🚀 GTX 1060 OPTIMIZED: 256 threads per block for better GPU utilization
    const int threads_per_block = 256;
    int blocks = (num_blocks + threads_per_block - 1) / threads_per_block;
    rinhash_cuda_kernel_batch<<<blocks, threads_per_block>>>(
        d_headers, block_header_len, d_outputs, num_blocks, d_memories, m_cost
    );
    hipDeviceSynchronize();
    check_cuda("rinhash_cuda_kernel_batch");

    err = hipMemcpy(outputs, d_outputs, outputs_size, hipMemcpyDeviceToHost);
    if (err != hipSuccess) { fprintf(stderr, "CUDA: copy output batch fail\n"); }

    hipFree(d_headers);
    hipFree(d_outputs);
    hipFree(d_memories);
}

// Helper function to convert a block header to bytes
extern "C" void blockheader_to_bytes(
    const uint32_t* version,
    const uint32_t* prev_block,
    const uint32_t* merkle_root,
    const uint32_t* timestamp,
    const uint32_t* bits,
    const uint32_t* nonce,
    uint8_t* output,
    size_t* output_len
) {
    size_t offset = 0;
    memcpy(output + offset, version, 4); offset += 4;
    memcpy(output + offset, prev_block, 32); offset += 32;
    memcpy(output + offset, merkle_root, 32); offset += 32;
    memcpy(output + offset, timestamp, 4); offset += 4;
    memcpy(output + offset, bits, 4); offset += 4;
    memcpy(output + offset, nonce, 4); offset += 4;
    *output_len = offset;
}

// Main RinHash function that would be called from outside
extern "C" void RinHash(
    const uint32_t* version,
    const uint32_t* prev_block,
    const uint32_t* merkle_root,
    const uint32_t* timestamp,
    const uint32_t* bits,
    const uint32_t* nonce,
    uint8_t* output
) {
    uint8_t block_header[80]; // Standard block header size
    size_t block_header_len;
    blockheader_to_bytes(
        version,
        prev_block,
        merkle_root,
        timestamp,
        bits,
        nonce,
        block_header,
        &block_header_len
    );
    rinhash_cuda(block_header, block_header_len, output);
}

bool is_better(uint8_t* hash1, uint8_t* hash2) {
    for (int i = 7; i >= 0; i--) {
        uint32_t h1 = ((uint32_t)hash1[i*4 + 0]) |
                      ((uint32_t)hash1[i*4 + 1] << 8) |
                      ((uint32_t)hash1[i*4 + 2] << 16) |
                      ((uint32_t)hash1[i*4 + 3] << 24);
        uint32_t h2 = ((uint32_t)hash2[i*4 + 0]) |
                      ((uint32_t)hash2[i*4 + 1] << 8) |
                      ((uint32_t)hash2[i*4 + 2] << 16) |
                      ((uint32_t)hash2[i*4 + 3] << 24);
        if (h1 < h2) return true;
        if (h1 > h2) return false;
    }
    return false; // equal
}

// 🚀 OPTIMIZED: Enhanced mining function with target-aware early termination
extern "C" void RinHash_mine_optimized(
    const uint32_t* work_data,
    uint32_t nonce_offset,
    uint32_t start_nonce,
    uint32_t num_nonces,
    uint32_t* target,           // 8 x uint32_t target  
    uint32_t* found_nonce,
    uint8_t* target_hash,
    uint8_t* best_hash,
    uint32_t* solution_found    // 1 if target was met
) {
    const size_t block_header_len = 80;
    if (num_nonces > MAX_BATCH_BLOCKS) {
        fprintf(stderr, "Mining batch too large (max %u)\n", MAX_BATCH_BLOCKS);
        return;
    }
    
    std::vector<uint8_t> block_headers(block_header_len * num_nonces);
    std::vector<uint8_t> hashes(32 * num_nonces);
    uint32_t solution_nonce = 0;

    // Prepare block headers with different nonces
    for (uint32_t i = 0; i < num_nonces; i++) {
        uint32_t current_nonce = start_nonce + i;
        uint32_t work_data_copy[20];
        memcpy(work_data_copy, work_data, 80);
        work_data_copy[nonce_offset] = current_nonce;
        memcpy(&block_headers[i * block_header_len], work_data_copy, 80);
    }

    // Use optimized kernel with target checking
    rinhash_cuda_batch_optimized(
        block_headers.data(), block_header_len, hashes.data(), num_nonces,
        target, solution_found, &solution_nonce
    );

    if (*solution_found) {
        // Solution found! Extract the winning hash
        *found_nonce = solution_nonce;
        uint32_t winner_index = solution_nonce - start_nonce;
        if (winner_index < num_nonces) {
            memcpy(best_hash, hashes.data() + winner_index * 32, 32);
        }
    } else {
        // No solution, find best hash
        memcpy(best_hash, hashes.data(), 32);
        *found_nonce = start_nonce;
        for (uint32_t i = 1; i < num_nonces; i++) {
            uint8_t* current_hash = hashes.data() + i * 32;
            if (is_better(current_hash, best_hash)) {
                memcpy(best_hash, current_hash, 32);
                *found_nonce = start_nonce + i;
            }
        }
    }
}

// Legacy mining function (kept for compatibility)
extern "C" void RinHash_mine(
    const uint32_t* work_data,
    uint32_t nonce_offset,
    uint32_t start_nonce,
    uint32_t num_nonces,
    uint32_t* found_nonce,
    uint8_t* target_hash,
    uint8_t* best_hash
) {
    const size_t block_header_len = 80;
    if (num_nonces > MAX_BATCH_BLOCKS) {
        fprintf(stderr, "Mining batch too large (max %u)\n", MAX_BATCH_BLOCKS);
        return;
    }
    std::vector<uint8_t> block_headers(block_header_len * num_nonces);
    std::vector<uint8_t> hashes(32 * num_nonces);

    // Prepare block headers with different nonces
    for (uint32_t i = 0; i < num_nonces; i++) {
        uint32_t current_nonce = start_nonce + i;
        uint32_t work_data_copy[20];
        memcpy(work_data_copy, work_data, 80);
        work_data_copy[nonce_offset] = current_nonce;
        memcpy(&block_headers[i * block_header_len], work_data_copy, 80);
    }

    // Calculate hashes for all nonces
    rinhash_cuda_batch(block_headers.data(), block_header_len, hashes.data(), num_nonces);

    // Initialize best_hash with the first hash
    memcpy(best_hash, hashes.data(), 32);
    *found_nonce = start_nonce;
    for (uint32_t i = 1; i < num_nonces; i++) {
        uint8_t* current_hash = hashes.data() + i * 32;
        if (is_better(current_hash, best_hash)) {
            memcpy(best_hash, current_hash, 32);
            *found_nonce = start_nonce + i;
        }
    }
}

// MWEB-enhanced hash function
extern "C" void RinHash_MWEB(
    const uint32_t* version,
    const uint32_t* prev_block,
    const uint32_t* merkle_root,
    const uint32_t* timestamp,
    const uint32_t* bits,
    const uint32_t* nonce,
    const uint8_t* mweb_hash,      // MWEB extension block hash (32 bytes)
    uint8_t mweb_present,          // 1 if MWEB data is present, 0 otherwise
    uint8_t* output
) {
    // Use standard RinHash as base
    RinHash(version, prev_block, merkle_root, timestamp, bits, nonce, output);
    
    // If MWEB is present, XOR with MWEB hash for additional mixing
    if (mweb_present && mweb_hash) {
        for (int i = 0; i < 32; i++) {
            output[i] ^= mweb_hash[i % 32];
        }
    }
}

// MWEB-enhanced mining function  
extern "C" void RinHash_MWEB_mine(
    const uint32_t* work_data,
    uint32_t nonce_offset,
    uint32_t start_nonce,
    uint32_t num_nonces,
    uint32_t* found_nonce,
    uint8_t* target_hash,
    uint8_t* best_hash,
    const uint8_t* mweb_hash,      // MWEB extension block hash
    uint8_t mweb_present           // MWEB presence flag
) {
    // Use regular mining then post-process if MWEB is present
    RinHash_mine(work_data, nonce_offset, start_nonce, num_nonces, found_nonce, target_hash, best_hash);
    
    // Apply MWEB mixing to best hash if present
    if (mweb_present && mweb_hash) {
        for (int i = 0; i < 32; i++) {
            best_hash[i] ^= mweb_hash[i % 32];
        }
    }
}
